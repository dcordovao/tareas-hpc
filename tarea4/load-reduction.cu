
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <iostream>
#include <fstream>

#define WIDTH 8192
#define LENGHT 8192
#define N_PARTICLES 5000
#define INF 999999.999
#define RADIO 100
#define CELLS_FOR_THREAD 8

using namespace std;

// __constant__ float x_part_dev[N_PARTICLES];
// __constant__ float y_part_dev[N_PARTICLES];

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of cudaError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      cout << hipGetErrorString(error) << endl; \
    } \
  } while (0)


__device__ float dist(float x1, float y1, float x2, float y2)
{
  float dist;
  dist = (x2-x1)*(x2-x1) + (y2-y1)*(y2-y1);
  //dist = sqrtf(powf(x2-x1, 2) + powf(y2-y1, 2));
  if(dist != 0) return 1/dist;
  else return -1;
}

__global__ void charge(float l, float *map,float *X,float *Y)
{
  
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  float rowParticle,colParticle,rowCell,colCell;

  for (int i = idx*CELLS_FOR_THREAD; i<idx*CELLS_FOR_THREAD+CELLS_FOR_THREAD; i++)
  {
    if (i<l)
    {
      for (size_t j = 0; j < N_PARTICLES; j++) {
        rowParticle = Y[j];
        colParticle = X[j];
        rowCell = (i / WIDTH);
        colCell = (i % WIDTH);
        //float distancia = rowCell-colCell;
        float distancia = 1;//(dist(rowParticle,colParticle,rowCell,colCell);
        if (distancia != -1) {
          map[i] += distancia;
        }
      }
      //map[i] = 1;
    }
  }
}

__global__
void chargeWithRadio(int l, float *map,float *X,float *Y)
{
  float d;
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int rowPartcile,colParticle,rowCell,colCell;

  if (idx < l)
  {
    for (size_t i = 0; i < N_PARTICLES; i++) {
      rowPartcile = Y[i];
      colParticle = X[i];
      rowCell = (idx / WIDTH)+1;
      colCell = (idx % WIDTH)+1;
      d = dist(rowPartcile,colParticle,rowCell,colCell);
      map[idx] += (d<RADIO)?d:0.0;
    }
  }
}

__global__
void minReduction(float *in, float *out)
{
  __shared__ float sharedData[256];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + tid; // blockSize = 256
  sharedData[tid] = in[i] + in[i+blockDim.x];
  __syncthreads();

  for (unsigned int  s = blockDim.x/2; s>32; s>>=1) {
    if(tid<s)
    {
      sharedData[tid] = (sharedData[tid]<sharedData[tid+s])?sharedData[tid]:sharedData[tid+s];
    }
    __syncthreads();
  }

  if (tid < 32)
  {
    sharedData[tid] = (sharedData[tid]<sharedData[tid+32])?sharedData[tid]:sharedData[tid+32];
    sharedData[tid] = (sharedData[tid]<sharedData[tid+16])?sharedData[tid]:sharedData[tid+16];
    sharedData[tid] = (sharedData[tid]<sharedData[tid+8])?sharedData[tid]:sharedData[tid+8];
    sharedData[tid] = (sharedData[tid]<sharedData[tid+4])?sharedData[tid]:sharedData[tid+4];
    sharedData[tid] = (sharedData[tid]<sharedData[tid+2])?sharedData[tid]:sharedData[tid+2];
    sharedData[tid] = (sharedData[tid]<sharedData[tid+1])?sharedData[tid]:sharedData[tid+1];
  }

  if(tid==0)
  {
    out[blockIdx.x] = sharedData[0];
  }
}

int main(int argc, char *argv[]){
  // Load data
  string input_file_name;

  if (argc > 1) {
		input_file_name = argv[1];
	} else {
		cout << "faltó un argumento" << endl;
		exit(0);
	}

	ifstream infile;
	cout << "Reading: " <<  input_file_name.c_str() << endl;
	infile.open(input_file_name.c_str());

  int nP;

	float *x_part, *y_part;

  infile >> nP;
  cout << "nP: "<<nP << endl;

  x_part = (float *)malloc(nP * sizeof(float));
	y_part = (float *)malloc(nP * sizeof(float));

  for (int i = 0; i<nP; i++) {
		infile >> x_part[i] >> y_part[i];
	}

  // Get memory for structures
  float *cells, *d_cells,*outData,*out2,*out3,y[4];
  float *x_part_dev, *y_part_dev;
  cells = (float*)malloc(WIDTH*LENGHT*sizeof(float));
  

  // Initialization grid with 0
  for (int i = 0; i < WIDTH*LENGHT; i++) {
    cells[i] = 0.0;
  }

  // Define sizes of GPU
  int blockSize = 256; // # threads
  int gridSize = ((WIDTH*LENGHT)/256)/CELLS_FOR_THREAD; // # blocks

  cout << "gridSize: " << gridSize << endl; 
  // Get memory in GPU for structures

  // data for charge function
  //cudaMalloc(&x_dev, nP * sizeof(float)); // X cord for particles
  //cudaMalloc(&y_dev, nP * sizeof(float)); // Y cord for particles
  CUDA_CHECK(hipMalloc(&d_cells, WIDTH*LENGHT*sizeof(float))); // 1D array representation for grid 2D
  CUDA_CHECK(hipMalloc(&x_part_dev, N_PARTICLES*sizeof(float)));
  CUDA_CHECK(hipMalloc(&y_part_dev, N_PARTICLES*sizeof(float)));

  // data for reduction function
  CUDA_CHECK(hipMalloc(&outData, gridSize*sizeof(float)));
  CUDA_CHECK(hipMalloc(&out2, (gridSize/blockSize)*sizeof(float)));
  CUDA_CHECK(hipMalloc(&out3, ((gridSize/blockSize)/blockSize)*sizeof(float)));

  // Copy data from CPU to GPU
  CUDA_CHECK(hipMemcpy(d_cells, cells, WIDTH*LENGHT*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(x_part_dev, x_part, N_PARTICLES * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(y_part_dev, y_part, N_PARTICLES * sizeof(float), hipMemcpyHostToDevice));
  //cudaMemcpy(x_dev, &x_part,  nP * sizeof(float), cudaMemcpyHostToDevice);
  //cudaMemcpy(y_dev, &y_part,  nP * sizeof(float), cudaMemcpyHostToDevice);


  hipEvent_t ct1, ct2;
  float dt, dt2;

  // time before kernel
  hipEventCreate(&ct1);
  hipEventCreate(&ct2);
  hipEventRecord(ct1);

  // Charge grid
  charge<<<gridSize,blockSize>>>(WIDTH*LENGHT, d_cells, x_part_dev, y_part_dev); 
  hipDeviceSynchronize();

  //Time after charge kernel
  hipEventRecord(ct2);
  hipEventSynchronize(ct2);
  hipEventElapsedTime(&dt, ct1, ct2);
  float time1 = dt;

  std::cout << "Time GPU computing cells charges: " << time1 << "[ms]" << std::endl;

  CUDA_CHECK(hipMemcpy(cells, d_cells, WIDTH*LENGHT*sizeof(float), hipMemcpyDeviceToHost));
  hipDeviceSynchronize();

  
  // check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
  }

  for (size_t i = 0; i < 100; i++) {
    cout << cells[i] << ' ';
  }
  
  cout << endl;
  float suma = 0;
  for (int i = 0; i < WIDTH*LENGHT; i++) {
    if (cells[i] == 0)
    {
      cout << "i: " << i << " = 0"<< endl;
      break;
    }
    suma += cells[i];
  }
  cout << "Suma: " << suma << endl;

  cout << "\n \n primera parte exitosa (?)" << endl;

  // time before kernel min
  hipEventCreate(&ct1);
  hipEventCreate(&ct2);
  hipEventRecord(ct1);

  // Search min load
  minReduction<<<gridSize,blockSize>>>(d_cells,outData); // First reduction 8192*8192 -> (8192*8192+255)/ 256 = 262.144
  hipDeviceSynchronize();
  minReduction<<<gridSize/blockSize,blockSize>>>(outData,out2); // Second reduction 262.144 -> 262.144/256 = 1024
  hipDeviceSynchronize();
  minReduction<<<(gridSize/blockSize)/blockSize,blockSize>>>(out2,out3); // Third reduction 262.144 -> 4 :)
  hipDeviceSynchronize();

  //Time after min kernel
  hipEventRecord(ct2);
  hipEventSynchronize(ct2);
  hipEventElapsedTime(&dt2, ct1, ct2);
  float time2 = dt2;

  std::cout << "Time GPU computing minimum value: " << time2 << "[ms]" << std::endl;

  // check for errors
  error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
  }

  // Escribiendo resultado en archivo
  ofstream times_file;
  times_file.open("results_tarea_4_2.txt", ios_base::app);
  times_file << input_file_name.c_str() << endl;
  times_file << "Tiempo en charge kernel: "<< dt << "[ms]" << endl;
  times_file << "Tiempo en min kernel: "<< dt2 << "[ms]" << endl;

  hipMemcpy(y, out3, 4*sizeof(float), hipMemcpyDeviceToHost);

  int min=INF;
  // min load
  for (size_t i = 0; i < 4; i++) {
    min = (y[i]<min)?y[i]:min;
  }

  cout << min << endl;

  //cudaFree(x_dev);
  //cudaFree(y_dev);
  hipFree(d_cells);
  hipFree(outData);
  hipFree(out2);
  hipFree(out3);
  free(cells);
  free(x_part);
  free(y_part);

  return 0;
}
