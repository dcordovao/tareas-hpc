
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of cudaError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      cout << hipGetErrorString(error) << endl; \
    } \
  } while (0)

__global__
void add_vecs(int n, float *x, float *y, float *z)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<n)
  {
    z[i] = x[i]+y[i];
  }
}

int main(void)
{
  int N = 10;
  float *x, *y, *z, *d_x, *d_y, *d_z;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  z = (float*)malloc(N*sizeof(float));

  CUDA_CHECK(hipMalloc(&d_x, N*sizeof(float))); // 1D array representation for grid 2D
  CUDA_CHECK(hipMalloc(&d_y, N*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_z, N*sizeof(float)));
  
  for (int i=0; i<N; i++)
  {
    x[i] = i+1;
    y[i] = (i+1)*10; 
  }
  
  CUDA_CHECK(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_z, z, N*sizeof(float), hipMemcpyHostToDevice));

  int blockSize = 256; // # threads
  int gridSize = (N/blockSize)+1; // # blocks

  add_vecs<<<gridSize,blockSize>>>(N, d_x, d_y, d_z);
  hipDeviceSynchronize();

  // check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
  }

  CUDA_CHECK(hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; i++)
  {
    cout << z[i] << endl;
  }
}