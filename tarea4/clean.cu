#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iostream>
#include <fstream>

#define WIDTH 8192
#define LENGHT 8192
#define N_PARTICLES 5000
#define RADIO 100

using namespace std;

typedef struct
{
    float charge;
    int index;
} cell;

__constant__ float x_part_dev[N_PARTICLES];
__constant__ float y_part_dev[N_PARTICLES];

#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    if (error != hipSuccess) { \
      cout << hipGetErrorString(error) << endl; \
    } \
  } while (0)


__device__ float dist(float x1, float y1, float x2, float y2)
{
  float dist;
  dist = sqrtf(powf(x2-x1, 2) + powf(y2-y1, 2));
  if(dist != 0) return 1/dist;
  else return -1;
}



__global__ void charge(float l, cell *map)
{

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float rowParticle,colParticle,rowCell,colCell;

  if (i<l)
  {
    for (size_t j = 0; j < N_PARTICLES; j++) {
      rowParticle = y_part_dev[j];
      colParticle = x_part_dev[j];
      rowCell = (i / WIDTH);
      colCell = (i % WIDTH);
      float distancia = (dist(rowParticle,colParticle,rowCell,colCell));
      if (distancia != -1) {
        map[i].charge += distancia;
      }
    }
  }
}

__global__ void reduce0(cell *g_idata, cell *g_odata)
{
    extern __shared__ cell sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2)
    {
        if (tid % (2*s) == 0)
        {
            sdata[tid] = (sdata[tid].charge < sdata[tid + s].charge)? sdata[tid]: sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}

int main(int argc, char *argv[]){
  // Load data
  string input_file_name;

  if (argc > 1) {
		input_file_name = argv[1];
	} else {
		cout << "faltó un argumento" << endl;
		exit(0);
	}

	ifstream infile;
	cout << "Reading: " <<  input_file_name.c_str() << endl;
	infile.open(input_file_name.c_str());

  int nP;
	float *x_part, *y_part;

  infile >> nP;
  cout << "nP: "<<nP << endl;

  x_part = (float *)malloc(nP * sizeof(float));
	y_part = (float *)malloc(nP * sizeof(float));

  for (int i = 0; i<nP; i++) {
		infile >> x_part[i] >> y_part[i];
	}

  // Get memory for structures
  //float *cells, *d_cells,*outData,*out2,*out3,y[4];
  cell *cells, *d_cells, *dev_out, *dev_out2,*out;
  float *x_part_dev, *y_part_dev;
  //cells = (float*)malloc(WIDTH*LENGHT*sizeof(float));
  cells = (cell*)malloc(WIDTH*LENGHT*sizeof(cell));


  // Initialization grid with 0
  for (int i = 0; i < WIDTH*LENGHT; i++) {
    cells[i].charge = 0.0;
  }

  // Define sizes of GPU
  int blockSize = 256; // # threads
  int gridSize = ((WIDTH*LENGHT)/blockSize)+ ((WIDTH*LENGHT) % blockSize != 0); // # blocks
  int sharedBytes = blockSize*sizeof(cell);

  // Get memory in GPU for structures
  // data for charge function
  CUDA_CHECK(hipMalloc(&d_cells, WIDTH*LENGHT*sizeof(cell))); // 1D array representation for grid 2D
  //CUDA_CHECK(hipMalloc(&x_part_dev, N_PARTICLES*sizeof(float)));
  //CUDA_CHECK(hipMalloc(&y_part_dev, N_PARTICLES*sizeof(float)));

  // data for reduction function
  CUDA_CHECK(hipMalloc(&dev_out, gridSize*sizeof(cell)));
  CUDA_CHECK(hipMalloc(&dev_out2, (gridSize/blockSize)*sizeof(cell)));
  out = (cell*)malloc((gridSize/blockSize)*sizeof(cell));

  // Copy data from CPU to GPU
  CUDA_CHECK(hipMemcpy(d_cells, cells, WIDTH*LENGHT*sizeof(cell), hipMemcpyHostToDevice));
  //CUDA_CHECK(hipMemcpy(x_part_dev, x_part, N_PARTICLES * sizeof(float), hipMemcpyHostToDevice));
  //CUDA_CHECK(hipMemcpy(y_part_dev, y_part, N_PARTICLES * sizeof(float), hipMemcpyHostToDevice));
  hipMemcpyToSymbol(HIP_SYMBOL(x_part_dev), x_part, N_PARTICLES * sizeof(float))
  hipMemcpyToSymbol(HIP_SYMBOL(y_part_dev), y_part, N_PARTICLES * sizeof(float))

  hipEvent_t ct1, ct2;
  float dt, dt2;

  // time before kernel
  hipEventCreate(&ct1);
  hipEventCreate(&ct2);
  hipEventRecord(ct1);

  // Charge grid
  charge<<<gridSize,blockSize>>>(WIDTH*LENGHT, d_cells);
  hipDeviceSynchronize();

  //Time after charge kernel
  hipEventRecord(ct2);
  hipEventSynchronize(ct2);
  hipEventElapsedTime(&dt, ct1, ct2);
  float time1 = dt;

  std::cout << "Time GPU computing cells charges: " << time1 << "[ms]" << std::endl;

  //CUDA_CHECK(hipMemcpy(cells, d_cells, WIDTH*LENGHT*sizeof(float), hipMemcpyDeviceToHost));
  //hipDeviceSynchronize();


  // check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
  }

  // time before kernel min
  hipEventCreate(&ct1);
  hipEventCreate(&ct2);
  hipEventRecord(ct1);

  // Search min load
  reduce0<<<gridSize,blockSize,sharedBytes>>>(d_cells, dev_out);
  hipDeviceSynchronize();
  cout << "first reduction" << endl;
  reduce0<<<gridSize/blockSize,blockSize,sharedBytes>>>(dev_out, dev_out2);
  hipDeviceSynchronize();
  cout << "second reduction" << endl;
  CUDA_CHECK(hipMemcpy(out, dev_out2, gridSize/blockSize*sizeof(float), hipMemcpyDeviceToHost));
  // check for errors
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
  }

  //Time after min kernel
  hipEventRecord(ct2);
  hipEventSynchronize(ct2);
  hipEventElapsedTime(&dt2, ct1, ct2);
  float time2 = dt2;

  std::cout << "Time GPU computing minimum value: " << time2 << "[ms]" << std::endl;

  for (size_t i = 0; i < gridSize/blockSize; i++) {
    cout << out[i] << endl;
  }

  // Escribiendo resultado en archivo
  ofstream times_file;
  times_file.open("results_tarea_4_2.txt", ios_base::app);
  times_file << input_file_name.c_str() << endl;
  times_file << "Tiempo en charge kernel: "<< dt << "[ms]" << endl;
  times_file << "Tiempo en min kernel: "<< dt2 << "[ms]" << endl;

  hipFree(d_cells);
  hipFree(dev_out2);
  hipFree(dev_out);
  free(cells);
  free(out)
  free(x_part);
  free(y_part);

  return 0;
}
