
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}


int main() {
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();    
    printf("Hello World from CPU!\n");
    return 0;
}